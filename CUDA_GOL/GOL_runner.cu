#include "hip/hip_runtime.h"
#include "GOL_runner.cuh"

#define threadWidth 16
#define threadHeight 16


__global__ void stepper(bool* board, bool* newBoard, int* width, int* height) {
	int horizIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int vertIndex = blockIdx.y * blockDim.y + threadIdx.y;
	if (horizIndex < *width && vertIndex < *height) {
		int neighborCount = 0;
		int checkInd = horizIndex + 1;
		checkInd %= *width;
		checkInd = vertIndex * *width + checkInd;
		if (board[checkInd]) neighborCount++;
		checkInd = horizIndex - 1;
		checkInd += *width; checkInd %= *width;

	}
}

GOL::GOL(int width, int height, bool* data) {
	this->width = width;
	this->height = height;
	size = sizeof(bool) * width * height;
	board = (bool *)malloc(size);
	hipMemcpy(board, data, size, hipMemcpyHostToHost);
}

bool GOL::init() {
	size = sizeof(bool) * width * height;
	hipMalloc((void **)&d_board, size);
	hipMalloc((void**)&d_boardNew, size);

	size = sizeof(int);
	hipMalloc((void**)&d_width, size); hipMalloc((void**)&d_height, size);
	
	hipMemcpy(&d_board, &board, size, hipMemcpyHostToDevice);

	numBlockVert = height / threadHeight;
	if (height % threadHeight > 0) numBlockVert++;
	numBlockHoriz = width / threadWidth;
	if (height % threadWidth > 0) numBlockHoriz++;

	initialized = true;
}

bool GOL::step(bool show, bool* output) {
	dim3 dimGrid(numBlockHoriz, numBlockVert);
	dim3 dimBlock(threadWidth, threadHeight);

}