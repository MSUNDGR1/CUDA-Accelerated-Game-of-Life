#include "hip/hip_runtime.h"
#include "GOL_runner.cuh"

#define threadWidth 16
#define threadHeight 16


__device__ void horizCheck(int* numFill, bool* board, int* width, int* height, int* x, int* y) {
	int horizIndex, vertIndex, realIndex, count;
	vertIndex = (*y);
	if ((*x) + 1 == (*width)) { horizIndex = 0; }
	else { horizIndex = (*width) + 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;
	if ((*x) == 0) { horizIndex = (*width) - 1; }
	else { horizIndex = (*x) - 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;
	*numFill = count;
}


__device__ void vertCheck(int* numFill, bool* board, int* width, int* height, int* x, int* y) {
	int horizIndex, vertIndex, realIndex, count;
	horizIndex = (*x);
	if ((*y) + 1 == (*height)) { vertIndex = 0; }
	else { vertIndex = (*height) + 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;
	if ((*y) == 0) { vertIndex = (*height) - 1; }
	else { vertIndex = (*y) - 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;
	*numFill = count;
}

__device__ void cornerCheck(int* numFill, bool* board, int* width, int* height, int* x, int* y) {

}

__global__ void stepper(bool* board, bool* newBoard, int* width, int* height) {
	int horizIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int vertIndex = blockIdx.y * blockDim.y + threadIdx.y;
	if (horizIndex < *width && vertIndex < *height) {
		int neighborCount = 0;
		int checkInd = horizIndex + 1;
		checkInd %= *width;
		checkInd = vertIndex * *width + checkInd;
		if (board[checkInd]) neighborCount++;
		checkInd = horizIndex - 1;
		checkInd += *width; checkInd %= *width;

	}
}

GOL::GOL(int width, int height, bool* data) {
	this->width = width;
	this->height = height;
	size = sizeof(bool) * width * height;
	board = (bool *)malloc(size);
	hipMemcpy(board, data, size, hipMemcpyHostToHost);
}

bool GOL::init() {
	size = sizeof(bool) * width * height;
	hipMalloc((void **)&d_board, size);
	hipMalloc((void**)&d_boardNew, size);

	size = sizeof(int);
	hipMalloc((void**)&d_width, size); hipMalloc((void**)&d_height, size);
	
	hipMemcpy(&d_board, &board, size, hipMemcpyHostToDevice);

	numBlockVert = height / threadHeight;
	if (height % threadHeight > 0) numBlockVert++;
	numBlockHoriz = width / threadWidth;
	if (height % threadWidth > 0) numBlockHoriz++;

	initialized = true;
}

bool GOL::step(bool show, bool* output) {
	dim3 dimGrid(numBlockHoriz, numBlockVert);
	dim3 dimBlock(threadWidth, threadHeight);

}