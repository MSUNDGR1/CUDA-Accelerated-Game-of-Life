#include "GOL_runner.cuh"

GOL::GOL(int width, int height, bool* data) {
	this->width = new int(width);
	this->height = new int(height);
	int size = sizeof(bool) * width * height;
	this->board = (bool *)malloc(size);
	hipMemcpy(this->board, data, size, hipMemcpyHostToHost);

}