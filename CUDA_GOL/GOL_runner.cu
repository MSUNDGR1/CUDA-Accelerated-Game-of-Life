#include "hip/hip_runtime.h"
#include "GOL_runner.cuh"

#define threadWidth 16
#define threadHeight 16


__forceinline __device__ int horizCheck(bool* board, int* width, int* height, int* x, int* y) {
	int horizIndex, vertIndex, realIndex, count;
	vertIndex = (*y); count = 0;

	if ((*x) + 1 == (*width)) { horizIndex = 0; }
	else { horizIndex = (*width) + 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((*x) == 0) { horizIndex = (*width) - 1; }
	else { horizIndex = (*x) - 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	return count;
}


__forceinline __device__ int vertCheck( bool* board, int* width, int* height, int* x, int* y) {
	int horizIndex, vertIndex, realIndex, count;
	horizIndex = (*x); count = 0;

	if ((*y) + 1 == (*height)) { vertIndex = 0; }
	else { vertIndex = (*height) + 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((*y) == 0) { vertIndex = (*height) - 1; }
	else { vertIndex = (*y) - 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	return count;

}

__forceinline __device__ int cornerCheck(bool* board, int* width, int* height, int* x, int* y) {
	int horizIndex, vertIndex, realIndex, count;
	count = 0;

	if ((*y) + 1 == (*height)) { vertIndex = 0; }
	else { vertIndex = (*height) + 1; }
	if ((*x) + 1 == (*width)) { horizIndex = 0; }
	else { horizIndex = (*width) + 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((*x) == 0) { horizIndex = (*width) - 1; }
	else { horizIndex = (*x) - 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((*y) == 0) { vertIndex = (*height) - 1; }
	else { vertIndex = (*y) - 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((*x) + 1 == (*width)) { horizIndex = 0; }
	else { horizIndex = (*width) + 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	return count;
}

__global__ void stepper(bool* board, bool* newBoard, int* width, int* height) {
	int horizIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int vertIndex = blockIdx.y * blockDim.y + threadIdx.y;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (horizIndex < *width && vertIndex < *height) {
		int neighborCount = horizCheck(board, width, height, &x, &y);
		neighborCount += vertCheck(board, width, height, &x, &y);
		neighborCount += cornerCheck(board, width, height, &x, &y);

	}
}

GOL::GOL(int width, int height, bool* data) {
	this->width = width;
	this->height = height;
	size = sizeof(bool) * width * height;
	board = (bool *)malloc(size);
	hipMemcpy(board, data, size, hipMemcpyHostToHost);
}

bool GOL::init() {
	size = sizeof(bool) * width * height;
	hipMalloc((void **)&d_board, size);
	hipMalloc((void**)&d_boardNew, size);

	size = sizeof(int);
	hipMalloc((void**)&d_width, size); hipMalloc((void**)&d_height, size);
	
	hipMemcpy(&d_board, &board, size, hipMemcpyHostToDevice);

	numBlockVert = height / threadHeight;
	if (height % threadHeight > 0) numBlockVert++;
	numBlockHoriz = width / threadWidth;
	if (height % threadWidth > 0) numBlockHoriz++;

	initialized = true;
}

bool GOL::step(bool show, bool* output) {
	dim3 dimGrid(numBlockHoriz, numBlockVert);
	dim3 dimBlock(threadWidth, threadHeight);

}