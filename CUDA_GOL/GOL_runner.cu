#include "hip/hip_runtime.h"
#include "GOL_runner.cuh"

#define threadWidth 16
#define threadHeight 16


__forceinline __device__ int horizCheck(bool* board, int* width, int* height, int* x, int* y) {
	int horizIndex, vertIndex, realIndex, count;
	vertIndex = (*y); count = 0;

	if ((*x) + 1 == (*width)) { horizIndex = 0; }
	else { horizIndex = (*width) + 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((*x) == 0) { horizIndex = (*width) - 1; }
	else { horizIndex = (*x) - 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	return count;
}


__forceinline __device__ int vertCheck( bool* board, int* width, int* height, int* x, int* y) {
	int horizIndex, vertIndex, realIndex, count;
	horizIndex = (*x); count = 0;

	if ((*y) + 1 == (*height)) { vertIndex = 0; }
	else { vertIndex = (*height) + 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((*y) == 0) { vertIndex = (*height) - 1; }
	else { vertIndex = (*y) - 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	return count;

}

__forceinline __device__ int cornerCheck(bool* board, int* width, int* height, int* x, int* y) {
	int horizIndex, vertIndex, realIndex, count;
	count = 0;

	if ((*y) + 1 == (*height)) { vertIndex = 0; }
	else { vertIndex = (*height) + 1; }
	if ((*x) + 1 == (*width)) { horizIndex = 0; }
	else { horizIndex = (*width) + 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((*x) == 0) { horizIndex = (*width) - 1; }
	else { horizIndex = (*x) - 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((*y) == 0) { vertIndex = (*height) - 1; }
	else { vertIndex = (*y) - 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((*x) + 1 == (*width)) { horizIndex = 0; }
	else { horizIndex = (*width) + 1; }
	realIndex = (*width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	return count;
}

__global__ void stepper(bool* board, bool* newBoard, int* width, int* height) {
	int horizIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int vertIndex = blockIdx.y * blockDim.y + threadIdx.y;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (horizIndex < *width && vertIndex < *height) {
		int neighborCount = horizCheck(board, width, height, &x, &y);
		neighborCount += vertCheck(board, width, height, &x, &y);
		neighborCount += cornerCheck(board, width, height, &x, &y);
		int realIndex = vertIndex * (*width) + horizIndex;
		bool cellState = board[realIndex];
		if (cellState && neighborCount < 2) { newBoard[realIndex] = false; }
		else if (cellState && neighborCount > 3) { newBoard[realIndex] = true; }
		else if (!cellState && neighborCount == 3) { newBoard[realIndex] = true; }
	}
}

GOL::GOL(int width, int height, bool* data) {
	this->width = width;
	this->height = height;
	size = sizeof(bool) * width * height;
	board = (bool *)malloc(size);
	//hipMemcpy(board, data, size, hipMemcpyHostToHost);
	hipMemcpy(board, data, size, hipMemcpyHostToHost);
}

void GOL::init() {
	hipMalloc((void **)&d_board, size);
	hipMalloc((void **)&d_boardNew, size);

	size = sizeof(int);
	hipMalloc((void**)&d_width, size); hipMalloc((void**)&d_height, size);
	
	hipMemcpy(&d_board, &board, size, hipMemcpyHostToDevice);

	numBlockVert = height / threadHeight;
	if (height % threadHeight > 0) numBlockVert++;
	numBlockHoriz = width / threadWidth;
	if (height % threadWidth > 0) numBlockHoriz++;
}

void GOL::step() {
	dim3 dimGrid(numBlockHoriz, numBlockVert);
	dim3 dimBlock(threadWidth, threadHeight);
	stepper << <dimGrid, dimBlock >> > (&d_board, &d_boardNew, d_width, d_height);
	hipMemcpy(&d_board, &d_boardNew, size, hipMemcpyDeviceToDevice);
	hipMemcpy(board, &d_board, size, hipMemcpyDeviceToHost);
}

GOL::~GOL() {
	hipFree(&d_board); hipFree(&d_boardNew);
	hipFree(&d_width); hipFree(&d_height);
}