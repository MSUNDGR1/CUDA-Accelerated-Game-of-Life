#include "hip/hip_runtime.h"
#include "GOL_runner.cuh"
#include <stdio.h>
#define threadWidth 16
#define threadHeight 16


 __inline __device__ int horizCheck(bool* board, int width, int height, int x, int y) {
	int horizIndex, vertIndex, realIndex, count;
	vertIndex = (y); count = 0;

	if ((x) + 1 == (width)) { horizIndex = 0; }
	else { horizIndex = (width) + 1; }
	realIndex = (width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((x) == 0) { horizIndex = (width) - 1; }
	else { horizIndex = (x) - 1; }
	realIndex = (width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	return count;
}


__inline __device__ int vertCheck( bool* board, int width, int height, int x, int y) {
	int horizIndex, vertIndex, realIndex, count;
	horizIndex = (x); count = 0;

	if ((y) + 1 == (height)) { vertIndex = 0; }
	else { vertIndex = (height) + 1; }
	realIndex = (width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((y) == 0) { vertIndex = (height) - 1; }
	else { vertIndex = (y) - 1; }
	realIndex = (width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	return count;

}

__inline __device__ int cornerCheck(bool* board, int width, int height, int x, int y) {
	int horizIndex, vertIndex, realIndex, count;
	count = 0;

	if ((y) + 1 == (height)) { vertIndex = 0; }
	else { vertIndex = (height) + 1; }
	if ((x) + 1 == (width)) { horizIndex = 0; }
	else { horizIndex = (width) + 1; }
	realIndex = (width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((x) == 0) { horizIndex = (width) - 1; }
	else { horizIndex = (x) - 1; }
	realIndex = (width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((y) == 0) { vertIndex = (height) - 1; }
	else { vertIndex = (y) - 1; }
	realIndex = (width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	if ((x) + 1 == (width)) { horizIndex = 0; }
	else { horizIndex = (width) + 1; }
	realIndex = (width) * vertIndex + horizIndex;
	if (board[realIndex]) count++;

	return count;
}

 __global__ void stepper(bool* board, bool* newBoard, int* width, int* height) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	

	if (x < (*width) && y < (*height)) {
		int count = 0; int horizIndex, vertIndex, realIndex;

		vertIndex = (y);

		if ((x)+1 == (*width)) { horizIndex = 0; }
		else { horizIndex = (*width)+1; }
		realIndex = (*width)*vertIndex + horizIndex;
		if (board[realIndex]) count++;

		if ((x) == 0) { horizIndex = (*width)-1; }
		else { horizIndex = (x)-1; }
		realIndex = (*width)*vertIndex + horizIndex;
		if (board[realIndex]) count++;

		horizIndex = (x);

		if ((y)+1 == (*height)) { vertIndex = 0; }
		else { vertIndex = (*height)+1; }
		realIndex = (*width)*vertIndex + horizIndex;
		if (board[realIndex]) count++;

		if ((y) == 0) { vertIndex = (*height)-1; }
		else { vertIndex = (y)-1; }
		realIndex = (*width)*vertIndex + horizIndex;
		if (board[realIndex]) count++;

		if ((y)+1 == (*height)) { vertIndex = 0; }
		else { vertIndex = (*height)+1; }
		if ((x)+1 == (*width)) { horizIndex = 0; }
		else { horizIndex = (*width)+1; }
		realIndex = (*width)*vertIndex + horizIndex;
		if (board[realIndex]) count++;

		if ((x) == 0) { horizIndex = (*width)-1; }
		else { horizIndex = (x)-1; }
		realIndex = (*width)*vertIndex + horizIndex;
		if (board[realIndex]) count++;

		if ((y) == 0) { vertIndex = (*height)-1; }
		else { vertIndex = (y)-1; }
		realIndex = (*width)*vertIndex + horizIndex;
		if (board[realIndex]) count++;

		if ((x)+1 == (*width)) { horizIndex = 0; }
		else { horizIndex = (*width)+1; }
		realIndex = (*width)*vertIndex + horizIndex;
		if (board[realIndex]) count++;


		/*int neighborCount = horizCheck(board, *width, *height, x, y);
		neighborCount += vertCheck(board, *width, *height, x, y);
		neighborCount += cornerCheck(board, *width, *height, x, y);*/
		/*if (x < 12 && x > 8) { if (y < 12 && y > 8) { printf("X: %d Y: %d NeighborCount: %d\n", x, y, neighborCount); } }*/
		 realIndex = y * (*width) + x;
		bool cellState = board[realIndex];
		if (cellState && count < 2) { newBoard[realIndex] = false; }
		else if (cellState && count > 3) { newBoard[realIndex] = true; }
		else if (!cellState && count == 3) { newBoard[realIndex] = true; }
	}
}

GOL::GOL(int width, int height, bool* data) {
	this->width = width;
	this->height = height;
	size = sizeof(bool) * width * height;
	board = (bool *)malloc(size);
	//hipMemcpy(board, data, size, hipMemcpyHostToHost);
	hipMemcpy(board, data, size, hipMemcpyHostToHost);
}

void GOL::init() {
	hipMalloc((void **)&d_board, size);
	hipMalloc((void **)&d_boardNew, size);

	size = sizeof(int);
	hipMalloc((void**)&d_width, size); hipMalloc((void**)&d_height, size);
	hipMemcpy(d_width, &width, size, hipMemcpyHostToDevice);
	hipMemcpy(d_height, &height, size, hipMemcpyHostToDevice);


	size = sizeof(bool) * width * height;
	hipMemcpy(&d_board, &board, size, hipMemcpyHostToDevice);

	numBlockVert = height / threadHeight;
	if (height % threadHeight > 0) numBlockVert++;
	numBlockHoriz = width / threadWidth;
	if (height % threadWidth > 0) numBlockHoriz++;
}

bool GOL::step() {
	dim3 dimGrid(numBlockHoriz, numBlockVert);
	dim3 dimBlock(threadWidth, threadHeight);
	stepper << <dimGrid, dimBlock >> > (&d_board, &d_boardNew, d_width, d_height);
	
	hipMemcpy(&d_board, &d_boardNew, size, hipMemcpyDeviceToDevice);
	hipMemcpy(board, &d_board, size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	return true;
}

GOL::~GOL() {
	hipFree(&d_board); hipFree(&d_boardNew);
	hipFree(&d_width); hipFree(&d_height);
}