#include "GOL_runner.cuh"

#define threadWidth 16
#define threadHeight 16




GOL::GOL(int width, int height, bool* data) {
	this->width = width;
	this->height = height;
	int size = sizeof(bool) * width * height;
	this->board = (bool *)malloc(size);
	hipMemcpy(this->board, data, size, hipMemcpyHostToHost);
}

bool GOL::init() {
	size = sizeof(bool) * width * height;
	hipMalloc((void **)&d_board, size);
	hipMalloc((void**)&d_boardNew, size);

	hipMemcpy(&d_board, &board, size, hipMemcpyHostToDevice);

	numBlockVert = height / threadHeight;
	if (height % threadHeight > 0) numBlockVert++;
	numBlockHoriz = width / threadWidth;
	if (height % threadWidth > 0) numBlockHoriz++;

	initialized = true;
}

bool GOL::step(bool show, bool* output) {
	dim3 dimGrid(numBlockHoriz, numBlockVert);
	dim3 dimBlock(threadWidth, threadHeight);

}